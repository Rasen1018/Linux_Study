
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <vector>
#include <iostream>

/* Linux File in Dir find include */
#include <sys/types.h>
#include <dirent.h>
#include <error.h>

using namespace std;

typedef unsigned short ushort;

void dirDarkfile();		// ������ ����� Dark ���丮 �� ��� raw������ darkfn ���Ϳ� pushback �ϴ� �Լ� 
void dirGainfile();		// ������ ����� Gain ���丮 �� ��� raw������ gainfn ���Ϳ� pushback �ϴ� �Լ� 
void darkMap();			// darkfn�� ��հ����� darkMap.raw ���� �Լ�
void gainMap();			// gainfn�� ��հ����� darkMap.raw ���� �Լ�

vector<string> darkfn;	// Dark ���丮 �� raw���ϵ��� �����ϰ� �ִ� ����
vector<string> gainfn;	// Gain ���丮 �� raw���ϵ��� �����ϰ� �ִ� ����

void callibration();	// ���� �Լ�

__global__ void cudaFilesSum(ushort *inimg, float *averageimg,int width, int height)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	int offset = x +(y * width);

	if( x < width && y < height){
		*(averageimg+offset) += *(inimg + offset);
	}
}

__global__ void cudaPixelAvg(float *averageimg, ushort *outimg, int width, int height )
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	int offset = x +(y * width);

	if( x < width && y < height){
		*(outimg + offset) =  *(averageimg + offset) / 101;
	}
}

__global__ void cudaCalibration(ushort *darkMapImg, ushort *GainMapImg,
		ushort *MTF_VImg, double subGainAvg, 
		int width, int height, ushort *outimg)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	int offset = x +(y * width);

	if( x < width && y < height){
		*(outimg + offset) = 
			(ushort) ( abs( *(MTF_VImg + offset) - ( *(darkMapImg + offset) ) ) / (float)( *(GainMapImg + offset) )  * subGainAvg );
	}
}

int main()
{
	dirDarkfile();
	dirGainfile();
	darkMap();
	gainMap();

	callibration();

	printf("Success processing !!\n");
	return 0;
}

void darkMap() {
	FILE* infp, * outfp;
	char savefile[] = "./output1628x1628/DarkMap(1628).raw";

	ushort* inimg, * outimg;
	float* f_averageimg;

	int width = 1628;
	int height = 1628;
	int imageSize = width * height;

	inimg = (ushort*)malloc(sizeof(ushort) * imageSize);
	outimg = (ushort*)malloc(sizeof(ushort) * imageSize);
	f_averageimg = (float*)malloc(sizeof(float) * imageSize);

	memset(inimg, 0, sizeof(ushort) * imageSize);
	memset(outimg, 0, sizeof(ushort) * imageSize);
	memset(f_averageimg, 0, sizeof(float) * imageSize);

	float *d_averageimg;
	hipMalloc(&d_averageimg, sizeof(float) * imageSize);
	hipMemset(d_averageimg, 0, sizeof(float) * imageSize);

	const dim3 dimGrid((int)ceil((width/4)), (int)ceil((height)/4));
	const dim3 dimBlock(4, 4);

	vector<string>::iterator iter;
	iter = darkfn.begin();
	for (iter = darkfn.begin(); iter != darkfn.end(); iter++) {
		memset(inimg, 0, sizeof(ushort) * imageSize);
		char path[100] = "./S1_1628x1628/Dark/";
		string file = path + *iter;

		//cout << file << endl;	// ���� fopen Ȯ��

		if ((infp = fopen(file.c_str(), "rb")) == NULL) {
			printf("%d No such file or folder\n", __LINE__);
			return;
		}

		fread(inimg, sizeof(ushort) * imageSize, 1, infp);

		/* cuda reset */
		ushort *d_inimg = NULL;

		hipMalloc(&d_inimg, sizeof(ushort) * imageSize);
		hipMemset(d_inimg, 0, sizeof(ushort) * imageSize);
		hipMemcpy(d_inimg, inimg, sizeof(ushort) * imageSize, hipMemcpyHostToDevice);


		cudaFilesSum<<<dimGrid, dimBlock>>>(d_inimg, d_averageimg, width, height);
		hipFree(d_inimg);

		fclose(infp);
	}

	ushort *d_outimg;
	hipMalloc(&d_outimg, sizeof(ushort) * imageSize);
	hipMemset(d_outimg, 0, sizeof(ushort) * imageSize);

	cudaPixelAvg<<<dimGrid, dimBlock>>>(d_averageimg ,d_outimg, width, height);

	hipMemcpy(outimg, d_outimg, sizeof(ushort) * imageSize, hipMemcpyDeviceToHost);

	hipFree(d_outimg);
	hipFree(d_averageimg);

	if ((outfp = fopen(savefile, "wb")) == NULL) {
		printf("%d No such file or folder\n", __LINE__);
		return;
	}

	fwrite(outimg, sizeof(ushort) * imageSize, 1, outfp);

	free(inimg);
	free(outimg);
	free(f_averageimg);
	fclose(outfp);
}


void gainMap() {
	FILE* infp, * outfp;
	char savefile[] = "./output1628x1628/GainMap(1628).raw";

	ushort* inimg, * outimg;
	float* f_averageimg;

	int width = 1628;
	int height = 1628;
	int imageSize = width * height;

	inimg = (ushort*)malloc(sizeof(ushort) * imageSize);
	outimg = (ushort*)malloc(sizeof(ushort) * imageSize);
	f_averageimg = (float*)malloc(sizeof(float) * imageSize);

	memset(inimg, 0, sizeof(ushort) * imageSize);
	memset(outimg, 0, sizeof(ushort) * imageSize);
	memset(f_averageimg, 0, sizeof(float) * imageSize);

	float *d_averageimg;
	hipMalloc(&d_averageimg, sizeof(float) * imageSize);
	hipMemset(d_averageimg, 0, sizeof(float) * imageSize);

	const dim3 dimGrid((int)ceil((width/4)), (int)ceil((height)/4));
	const dim3 dimBlock(4, 4);

	vector<string>::iterator iter;
	iter = gainfn.begin();
	for (iter = gainfn.begin(); iter != gainfn.end(); iter++) {
		memset(inimg, 0, sizeof(ushort) * imageSize);
		char path[100] = "./S1_1628x1628/Gain/";
		string file = path + *iter;

		//cout << file << endl;	// ���� fopen Ȯ��

		if ((infp = fopen(file.c_str(), "rb")) == NULL) {
			printf("%d No such file or folder\n", __LINE__);
			return;
		}

		fread(inimg, sizeof(ushort) * imageSize, 1, infp);
		fclose(infp);

		ushort *d_inimg = NULL;
		hipMalloc(&d_inimg, sizeof(ushort) * imageSize);
		hipMemset(d_inimg, 0, sizeof(ushort) * imageSize);
		hipMemcpy(d_inimg, inimg, sizeof(ushort) * imageSize, hipMemcpyHostToDevice);


		cudaFilesSum<<<dimGrid, dimBlock>>>(d_inimg, d_averageimg, width, height);
		hipFree(d_inimg);

	}

	ushort *d_outimg;
	hipMalloc(&d_outimg, sizeof(ushort) * imageSize);
	hipMemset(d_outimg, 0, sizeof(ushort) * imageSize);

	cudaPixelAvg<<<dimGrid, dimBlock>>>(d_averageimg ,d_outimg, width, height);

	hipMemcpy(outimg, d_outimg, sizeof(ushort) * imageSize, hipMemcpyDeviceToHost);

	hipFree(d_outimg);
	hipFree(d_averageimg);

	if ((outfp = fopen(savefile, "wb")) == NULL) {
		printf("%d No such file or folder\n", __LINE__);
		return;
	}
	fwrite(outimg, sizeof(ushort) * imageSize, 1, outfp);

	free(inimg);
	free(outimg);
	free(f_averageimg);
	fclose(outfp);
}

void callibration() {
	FILE* GainMapFp, * darkMapFp, * MTF_VFp, *outfp;
	char savefile[] = "./output1628x1628/callibration(1628).raw";

	int width = 1628, height = 1628;
	int imageSize = width * height;
	int subImageSize = (width - 200) * (height - 200);
	int widthcnt = 0;
	double subGainSum = 0, subGainAvg = 0;


	ushort * darkMapImg, * GainMapImg,* MTF_VImg, *outimg;

	darkMapImg = (ushort*)malloc(sizeof(ushort) * imageSize);
	GainMapImg = (ushort*)malloc(sizeof(ushort) * imageSize);
	MTF_VImg = (ushort*)malloc(sizeof(ushort) * imageSize);
	outimg = (ushort*)malloc(sizeof(ushort) * imageSize);

	memset(darkMapImg, 0, sizeof(ushort) * imageSize);
	memset(GainMapImg, 0, sizeof(ushort) * imageSize);
	memset(MTF_VImg, 0, sizeof(ushort) * imageSize);
	memset(outimg, 0, sizeof(ushort) * imageSize);

	if ((darkMapFp = fopen("./output1628x1628/DarkMap(1628).raw", "rb")) == NULL) {
		printf("%d No such file or folder\n", __LINE__);
		return;
	}
	if ((GainMapFp = fopen("./output1628x1628/GainMap(1628).raw", "rb")) == NULL) {
		printf("%d No such file or folder\n", __LINE__);
		return;
	}
	if ((MTF_VFp = fopen("./S1_1628x1628/MTF_V.raw", "rb")) == NULL) {
		printf("%d No such file or folder\n", __LINE__);
		return;
	}

	fread(darkMapImg, sizeof(ushort) * imageSize, 1, darkMapFp);
	fread(GainMapImg, sizeof(ushort) * imageSize, 1, GainMapFp);
	fread(MTF_VImg, sizeof(ushort) * imageSize, 1, MTF_VFp);

	fclose(darkMapFp);
	fclose(GainMapFp);
	fclose(MTF_VFp);

	/* cuda reset */
	ushort *d_MTF_VImg, *d_darkMapImg, *d_GainMapImg, *d_outimg;

	const dim3 dimGrid((int)ceil((width/4)), (int)ceil((height)/4));
	const dim3 dimBlock(4, 4);
	
	hipMalloc(&d_darkMapImg, sizeof(ushort) * imageSize);
	hipMalloc(&d_GainMapImg, sizeof(ushort) * imageSize);
	hipMalloc(&d_MTF_VImg, sizeof(ushort) * imageSize);

	hipMemset(d_darkMapImg, 0, sizeof(ushort) * imageSize);
	hipMemset(d_GainMapImg, 0, sizeof(ushort) * imageSize);
	hipMemset(d_MTF_VImg, 0, sizeof(ushort) * imageSize);

	hipMemcpy(d_darkMapImg, darkMapImg, sizeof(ushort) * imageSize, hipMemcpyHostToDevice);
	hipMemcpy(d_GainMapImg, GainMapImg, sizeof(ushort) * imageSize, hipMemcpyHostToDevice);
	hipMemcpy(d_MTF_VImg, MTF_VImg, sizeof(ushort) * imageSize, hipMemcpyHostToDevice);

	hipMalloc(&d_outimg, sizeof(ushort) * imageSize);
	hipMemset(d_outimg, 0, sizeof(ushort) * imageSize);

	for (int i = 0; i < imageSize; i++) {
		widthcnt++;
		if (widthcnt == width) widthcnt = 0;

		if (width * 100 > i || width * (height - 100) < i) continue;
		if (widthcnt <= 100 || widthcnt > 1528) continue;

		subGainSum += GainMapImg[i];
	}

	subGainAvg = subGainSum / subImageSize;

	cudaCalibration<<<dimGrid, dimBlock>>>(d_darkMapImg, d_GainMapImg, d_MTF_VImg,
			subGainAvg, width, height, d_outimg);	

	hipMemcpy(outimg, d_outimg, sizeof(ushort) * imageSize, hipMemcpyDeviceToHost);

	hipFree(d_MTF_VImg);
	hipFree(d_darkMapImg);
	hipFree(d_GainMapImg);
	hipFree(d_outimg);

	if ((outfp = fopen(savefile, "wb")) == NULL) {
		printf("%d No such file or folder\n", __LINE__);
		return;
	}

	fwrite(outimg, sizeof(ushort) * imageSize, 1, outfp);

	fclose(outfp);

	free(GainMapImg);
	free(darkMapImg);
	free(MTF_VImg);
	free(outimg);
}

// dark 디렉토리 내 파일 찾기 함수
void dirDarkfile()
{
	DIR *dir;
	struct dirent *ent;
	dir = opendir ("./S1_1628x1628/Dark/");
	if (dir != NULL) {
		/* print all the files and directories within directory */
		while ((ent = readdir (dir)) != NULL) {
			string file = ent->d_name;
			if( file.find(".raw") ==string::npos ) continue;
			else{
				darkfn.push_back(ent->d_name);
			}
		}
		closedir (dir);
	} else {
		/* could not open directory */
		perror ("");
		return;
	}
}

// Gain 디렉토리 내 파일 찾기 함수
void dirGainfile()
{
	DIR *dir;
	struct dirent *ent;
	dir = opendir ("./S1_1628x1628/Gain/");
	if (dir != NULL) {
		/* print all the files and directories within directory */
		while ((ent = readdir (dir)) != NULL) {
			string file = ent->d_name;
			if( file.find(".raw") ==string::npos ) continue;
			else{
				gainfn.push_back(ent->d_name);
			}
		}
		closedir (dir);
	} else {
		/* could not open directory */
		perror ("");
		return;
	}
}
