
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel()
{
	printf("Hi GPU\n");
}

int main(int argc, char** argv)
{
	printf("---Hello CPU---\n");
	kernel<<<3,1>>>();
	hipDeviceReset();
	printf("---BYE CPU---\n");

	return 0;
}
