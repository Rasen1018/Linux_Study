#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h> /* USHRT_MAX 상수를 위해서 사용한다. */
#include <math.h>
#include <iostream>
#include "bmpHeader.h"
/* 이미지 데이터의 경계 검사를 위한 매크로 */
#define LIMIT_UBYTE(n) ((n)>UCHAR_MAX)?UCHAR_MAX:((n)<0)?0:(n)
#define widthbytes(bits) (((bits)+31)/32*4)
typedef unsigned char ubyte;

//Create a noise standard deviation
__global__ void convertToBlur(ubyte *inimg, ubyte *out, int width, int height, int elemSize) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int z = threadIdx.z + blockIdx.z * blockDim.z;

	int rowSize = width*elemSize;
	int offset = (x*elemSize+(y*rowSize));


	float offset_Average, offset_SD;

	float sum = 0.0, sum_SD = 0.0, mask_SD = 0.0;
	int wiener = 0.0;

	// inSide
	if ( (x > 0 && x < width-1) && (y >0 && y < height-1) ) {
		for(int i = -1; i < 2; i++) {
			for(int j = -1; j< 2; j ++){
				sum += inimg[(x+i)*elemSize +((y+j)*rowSize) +z];
			}
		}
		offset_Average = sum/9;		//In mask average
		sum = 0.0;
		//find a standard deviation value
		for(int i = -1; i < 2; i++) {
			for(int j = -1; j< 2; j ++){
				offset_SD =  pow( ( inimg[(x+i)*elemSize + ((y+j)*rowSize)+z] - offset_Average), 2)  / 9 ;
				sum_SD += offset_SD;
			}
		}
		mask_SD = sqrt(sum_SD);
		float o = pow(mask_SD, 2);
		//Input noise_deviation
		float v = pow(noise_deviation, 2);
		//USE wiener filter
		wiener = offset_Average + (1 + v/o) * inimg(rgb[offset+z] - offset_Average);
		out[offset +z] = LIMIT_UBYTE(wiener);
	}
	// OutSide
	else{
		int arr[9] = {0, };
		//LeftSide
		else if(x ==0){
			//LeftTopVertex
			if(y==0){
				arr[0] = arr[1] = arr[3] = arr[4] = inimg[(x*elemSize)+(y*rowSize)+z];
				arr[2] = arr[5] = inimg[(x*elemSize)+elemSize +(y*rowSize)+z];
				arr[6] = arr[7] = inimg[(x*elemSize)+((y+1)*rowSize)+z];
				arr[8] = inimg[(x*elemSize)+elemSize+((y+1)*rowSize)+z];	
			}
			//LeftDownVertex
			else if(y==height-1){
				arr[0] = arr[1] = inimg[(x*elemSize)+((y-1)*rowSize)+z];
				arr[2] = inimg[(x*elemSize)+elemSize+((y-1)*rowSize)+z];
				arr[3] = arr[6] = arr[7] = arr[4] = inimg[(x*elemSize)+(y*rowSize)+z];
				arr[8] = arr[5] = inimg[(x*elemSize)+elemSize+(y*rowSize)+z];
			}

			//LeftSide
			else{
				arr[0] = arr[1] = inimg[(x*elemSize)+((y-1)*rowSize)+z];
				arr[2] = inimg[(x*elemSize)+elemSize+((y-1)*rowSize)+z];
				arr[3] = arr[4] = inimg[(x*elemSize)+(y*rowSize)+z];
				arr[5] = inimg[(x*elemSize)+elemSize+(y*rowSize)+z];
				arr[6] = arr[7] = inimg[(x*elemSize)+((y+1)*rowSize)+z];
				arr[8] = inimg[(x*elemSize)+elemSize+((y+1)*rowSize)+z];
			}

		}
		//RightSide
		else if(x==width-1){
			//RightTopVertex
			if(y==0){
				arr[0] = arr[3] = inimg[(x*elemSize)-elemSize+(y*rowSize)+z];
				arr[1] = arr[2] = arr[5] = arr[4] = inimg[offset+z];
				arr[6] = inimg[(x*elemSize)-elemSize+((y-1)*rowSize)+z];
				arr[7] = arr[8] = inimg[(x*elemSize)+((y+1)*rowSize)+z];
			}
			//RightDownVertex
			else if(y==height-1){
				arr[0] = inimg[(x*elemSize)-elemSize+((y-1)*rowSize)+z];
				arr[1] = arr[2] = inimg[(x*elemSize)-elemSize+((y-1)*rowSize)+z];
				arr[3] = arr[6] = inimg[(x*elemSize)-elemSize+(y*rowSize)+z];
				arr[4] = arr[5] = arr[7] = arr[8] = inimg[offset+z];
			}
			//RightSide
			else{
				arr[0] = inimg[(x*elemSize)-elemSize+((y-1)*rowSize)+z];
				arr[1] = arr[2] = inimg[(x*elemSize)+((y-1)*rowSize)+z];
				arr[3] = inimg[(x*elemSize)-elemSize+(y*rowSize)+z];
				arr[4] = arr[5] = inimg[(x*elemSize)+(y*rowSize)+z];
				arr[6] = inimg[(x*elemSize)-elemSize+((y+1)*rowSize)+z];
				arr[7] = arr[8] = inimg[(x*elemSize)+((y+1)*rowSize)+z];
			}
		}
		//TopSide
		else if( y==0){
			if(x!=0 && x!=width-1){
				arr[0] = arr[3] = inimg[(x*elemSize)-elemSize+(y*rowSize)+z];
				arr[1] = arr[4] = inimg[offset+z];
				arr[2] = arr[5] = inimg[(x*elemSize)+elemSize+(y*rowSize)+z];
				arr[6] = inimg[(x*elemSize)-elemSize+((y+1)*rowSize)+z];
				arr[7] = inimg[(x*elemSize)+((y+1)*rowSize)+z];
				arr[8] = inimg[(x*elemSize)+elemSize+((y+1)*rowSize)+z];

			}
		}
		//BottomSide
		else if( y==height-1){
			if(x!=0 && x!=width-1){
				arr[0] = inimg[(x*elemSize)-elemSize+((y-1)*rowSize)+z];
				arr[1] = inimg[(x*elemSize)+((y-1)*rowSize)+z];
				arr[2] = inimg[(x*elemSize)+elemSize+((y-1)*rowSize)+z];
				arr[3] = arr[6] = inimg[(x*elemSize)-elemSize+(y*rowSize)+z];
				arr[4] = arr[7] = inimg[offset+z];
				arr[5] = arr[8] = inimg[(x*elemSize)-elemSize+(y*rowSize)+z];
			}
		}	

		int cnt=0;
		//find sum of AdjacentValue
		for(int i = -1; i < 2; i++) {
			for(int j = -1; j< 2; j ++){
				sum += arr[cnt++];
			}
		}

		offset_Average = sum/9;		//In mask average
		cnt = 0;

		//find a standard deviation value
		for(int i = -1; i < 2; i++) {
			for(int j = -1; j< 2; j ++){
				offset_SD =  pow( ( arr[cnt++] - offset_Average), 2)  / 9 ;
				sum_SD += offset_SD;
			}
		}
		mask_SD = sqrt(sum_SD);
		float o = pow(mask_SD, 2);
		//Input noise_deviation
		float v = pow(noise_deviation, 2);
		//USE wiener filter
		wiener = offset_Average + (1 + v/o) * abs(inimg[offset+z] - offset_Average);
		out[offset +z] = LIMIT_UBYTE(wiener);
	}

}

int main(int argc, char** argv)
{
	FILE* fp;
	BITMAPFILEHEADER bmpHeader; /* BMP FILE INFO */
	BITMAPINFOHEADER bmpInfoHeader; /* BMP IMAGE INFO */

	ubyte *inimg, *outimg;
	if(argc != 3) {
		fprintf(stderr, "usage : %s input.bmp output.bmp\n", argv[0]);
		return -1;
	}
	/***** read bmp *****/
	if((fp=fopen(argv[1], "rb")) == NULL) {
		fprintf(stderr, "Error : Failed to open file...₩n");
		return -1;
	}
	/* BITMAPFILEHEADER 구조체의 데이터 */
	fread(&bmpHeader, sizeof(BITMAPFILEHEADER), 1, fp);
	/* BITMAPINFOHEADER 구조체의 데이터 */
	fread(&bmpInfoHeader, sizeof(BITMAPINFOHEADER), 1, fp);
	/* 트루 컬러를 지원하면 변환할 수 없다. */
	if(bmpInfoHeader.biBitCount != 24) {
		perror("This image file doesn't supports 24bit color\n");
		fclose(fp);
		return -1;
	}

	int elemSize = bmpInfoHeader.biBitCount/8.;
	int stride = bmpInfoHeader.biWidth * elemSize;
	int imageSize = stride * bmpInfoHeader.biHeight;

	inimg = (ubyte*)malloc(sizeof(ubyte)*imageSize);
	outimg = (ubyte*)malloc(sizeof(ubyte)*imageSize);

	fread(inimg, sizeof(ubyte), imageSize, fp);
	fclose(fp);

	ubyte *d_inimg = NULL, *d_outimg = NULL;
	//allocate and initialize memory on device
	hipMalloc(&d_inimg, sizeof(ubyte) * imageSize);
	hipMalloc(&d_outimg, sizeof(ubyte) * imageSize);
	hipMemset(d_outimg, 0, sizeof(ubyte) * imageSize);
	//copy host rgb data array to device rgb data array
	hipMemcpy(d_inimg, inimg, sizeof(ubyte) * imageSize, hipMemcpyHostToDevice);

	//define block and grid dimensions
	const dim3 dimGrid((int)ceil((bmpInfoHeader.biWidth/32)), (int)ceil((bmpInfoHeader.biHeight)/4),1);
	const dim3 dimBlock(32, 4, elemSize);

	//execute cuda kernel
	convertToBlur<<<dimGrid, dimBlock>>>(d_inimg, d_outimg, bmpInfoHeader.biHeight, bmpInfoHeader.biWidth, elemSize);
	//copy computed gray data array from device to host
	hipMemcpy(outimg, d_outimg, sizeof(ubyte) * imageSize, hipMemcpyDeviceToHost);

	hipFree(d_outimg);
	hipFree(d_inimg);

	/***** write bmp *****/
	if((fp=fopen(argv[2], "wb"))==NULL) {
		fprintf(stderr, "Error : Failed to open file...₩n");
		return -1;
	}

	bmpInfoHeader.biBitCount = 24;
	bmpInfoHeader.SizeImage = imageSize;

	bmpHeader.bfSize = bmpInfoHeader.SizeImage;
	/* BITMAPFILEHEADER 구조체의 데이터 */
	fwrite(&bmpHeader, sizeof(BITMAPFILEHEADER), 1, fp);
	/* BITMAPINFOHEADER 구조체의 데이터 */
	fwrite(&bmpInfoHeader, sizeof(BITMAPINFOHEADER), 1, fp);

	fwrite(outimg, sizeof(ubyte), imageSize, fp);
	fclose(fp);
	free(inimg);
	free(outimg);

	printf("Success blur\n");
	return 0;
}
