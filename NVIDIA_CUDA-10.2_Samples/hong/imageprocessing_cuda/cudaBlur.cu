#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h> /* USHRT_MAX 상수를 위해서 사용한다. */
#include <math.h>
#include <iostream>
#include "bmpHeader.h"
/* 이미지 데이터의 경계 검사를 위한 매크로 */
#define LIMIT_UBYTE(n) ((n)>UCHAR_MAX)?UCHAR_MAX:((n)<0)?0:(n)
#define widthbytes(bits) (((bits)+31)/32*4)
typedef unsigned char ubyte;
//Cuda kernel for converting RGB image into a GreyScale image

__global__ void convertToBlur(ubyte *rgb, ubyte *out, int width, int height, int elemSize) {
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		int size = width*elemSize;
		int rgb_offset = (x*elemSize+(y*size));

		unsigned char arr[9]={0,};
		float blur[3][3] = { {1/9.0, 1/9.0, 1/9.0,},
				{1/9.0, 1/9.0, 1/9.0},
				{1/9.0, 1/9.0, 1/9.0} };

		for(int z = 0; z<elemSize; z++){
				// inSide
				float sum = 0.0;
				if ( (x > 0 && x < width-1) && (y >0 && y < height-1) ) {
						//else{
						for(int i = -1; i < 2; i++) {
								for(int j = -1; j < 2; j++) {
										sum += blur[i+1][j+1]*rgb[((x+i)+(y+j)*height)*elemSize+z];
								}
						}
						out[rgb_offset +z] = LIMIT_UBYTE(sum);
				}
				//LeftSide
				else if(x ==0){
						//LeftTopVertex
						if(y==0){
								arr[0] = arr[1] = arr[3] = arr[4] = rgb[(x*elemSize)+(y*size)+z];
								arr[2] = arr[5] = rgb[(x*elemSize)+elemSize +(y*size)+z];
								arr[6] = arr[7] = rgb[(x*elemSize)+((y+1)*size)+z];
								arr[8] = rgb[(x*elemSize)+elemSize+((y+1)*size)+z];	
						}
						//LeftDownVertex
						else if(y==height-1){
								arr[0] = arr[1] = rgb[(x*elemSize)+((y-1)*size)+z];
								arr[2] = rgb[(x*elemSize)+elemSize+((y-1)*size)+z];
								arr[3] = arr[6] = arr[7] = arr[4] = rgb[(x*elemSize)+(y*size)+z];
								arr[8] = arr[5] = rgb[(x*elemSize)+elemSize+(y*size)+z];
						}

						//LeftSide
						else{
								arr[0] = arr[1] = rgb[(x*elemSize)+((y-1)*size)+z];
								arr[2] = rgb[(x*elemSize)+elemSize+((y-1)*size)+z];
								arr[3] = arr[4] = rgb[(x*elemSize)+(y*size)+z];
								arr[5] = rgb[(x*elemSize)+elemSize+(y*size)+z];
								arr[6] = arr[7] = rgb[(x*elemSize)+((y+1)*size)+z];
								arr[8] = rgb[(x*elemSize)+elemSize+((y+1)*size)+z];
						}

				}
				//RightSide
				else if(x==width-1){
						//RightTopVertex
						if(y==0){
								arr[0] = arr[3] = rgb[(x*elemSize)-elemSize+(y*size)+z];
								arr[1] = arr[2] = arr[5] = arr[4] = rgb[rgb_offset+z];
								arr[6] = rgb[(x*elemSize)-elemSize+((y-1)*size)+z];
								arr[7] = arr[8] = rgb[(x*elemSize)+((y+1)*size)+z];
						}
						//RightDownVertex
						else if(y==height-1){
								arr[0] = rgb[(x*elemSize)-elemSize+((y-1)*size)+z];
								arr[1] = arr[2] = rgb[(x*elemSize)-elemSize+((y-1)*size)+z];
								arr[3] = arr[6] = rgb[(x*elemSize)-elemSize+(y*size)+z];
								arr[4] = arr[5] = arr[7] = arr[8] = rgb[rgb_offset+z];
						}
						//RightSide
						else{
								arr[0] = rgb[(x*elemSize)-elemSize+((y-1)*size)+z];
								arr[1] = arr[2] = rgb[(x*elemSize)+((y-1)*size)+z];
								arr[3] = rgb[(x*elemSize)-elemSize+(y*size)+z];
								arr[4] = arr[5] = rgb[(x*elemSize)+(y*size)+z];
								arr[6] = rgb[(x*elemSize)-elemSize+((y+1)*size)+z];
								arr[7] = arr[8] = rgb[(x*elemSize)+((y+1)*size)+z];
						}
				}
				//TopSide
				else if( y==0){
						if(x!=0 && x!=width-1){
								arr[0] = arr[3] = rgb[(x*elemSize)-elemSize+(y*size)+z];
								arr[1] = arr[4] = rgb[rgb_offset+z];
								arr[2] = arr[5] = rgb[(x*elemSize)+elemSize+(y*size)+z];
								arr[6] = rgb[(x*elemSize)-elemSize+((y+1)*size)+z];
								arr[7] = rgb[(x*elemSize)+((y+1)*size)+z];
								arr[8] = rgb[(x*elemSize)+elemSize+((y+1)*size)+z];

						}
				}
				//BottomSide
				else if( y==height-1){
						if(x!=0 && x!=width-1){
								arr[0] = rgb[(x*elemSize)-elemSize+((y-1)*size)+z];
								arr[1] = rgb[(x*elemSize)+((y-1)*size)+z];
								arr[2] = rgb[(x*elemSize)+elemSize+((y-1)*size)+z];
								arr[3] = arr[6] = rgb[(x*elemSize)-elemSize+(y*size)+z];
								arr[4] = arr[7] = rgb[rgb_offset+z];
								arr[5] = arr[8] = rgb[(x*elemSize)-elemSize+(y*size)+z];
						}
				}	
				int cnt=0;
				for(int i = -1; i < 2; i++) {
						for(int j = -1; j < 2; j++) {
								sum += blur[i+1][j+1]*arr[cnt++];
						}
				}
				out[rgb_offset+z] = LIMIT_UBYTE(sum);
				}//z for
		}

		int main(int argc, char** argv)
		{
				FILE* fp;
				BITMAPFILEHEADER bmpHeader; /* BMP FILE INFO */
				BITMAPINFOHEADER bmpInfoHeader; /* BMP IMAGE INFO */
				//RGBQUAD *palrgb;
				ubyte *inimg, *outimg;
				if(argc != 3) {
						fprintf(stderr, "usage : %s input.bmp output.bmp\n", argv[0]);
						return -1;
				}
				/***** read bmp *****/
				if((fp=fopen(argv[1], "rb")) == NULL) {
						fprintf(stderr, "Error : Failed to open file...₩n");
						return -1;
				}
				/* BITMAPFILEHEADER 구조체의 데이터 */
				fread(&bmpHeader, sizeof(BITMAPFILEHEADER), 1, fp);
				/* BITMAPINFOHEADER 구조체의 데이터 */
				fread(&bmpInfoHeader, sizeof(BITMAPINFOHEADER), 1, fp);
				/* 트루 컬러를 지원하면 변환할 수 없다. */
				if(bmpInfoHeader.biBitCount != 24) {
						perror("This image file doesn't supports 24bit color\n");
						fclose(fp);
						return -1;
				}

				int elemSize = bmpInfoHeader.biBitCount/8.;
				int stride = bmpInfoHeader.biWidth * elemSize;
				//widthbytes(bits) (((bits)+31)/32*4)
				int imageSize = stride * bmpInfoHeader.biHeight;

				inimg = (ubyte*)malloc(sizeof(ubyte)*imageSize);
				outimg = (ubyte*)malloc(sizeof(ubyte)*imageSize);

				fread(inimg, sizeof(ubyte), imageSize, fp);
				fclose(fp);

				ubyte *d_inimg = NULL, *d_outimg = NULL;
				//allocate and initialize memory on device
				hipMalloc(&d_inimg, sizeof(ubyte) * imageSize);
				hipMalloc(&d_outimg, sizeof(ubyte) * imageSize);
				hipMemset(d_outimg, 0, sizeof(ubyte) * imageSize);
				//copy host rgb data array to device rgb data array
				hipMemcpy(d_inimg, inimg, sizeof(ubyte) * imageSize, hipMemcpyHostToDevice);

				//define block and grid dimensions
				const dim3 dimGrid((int)ceil((bmpInfoHeader.biWidth/32)), (int)ceil((bmpInfoHeader.biHeight)/16));
				const dim3 dimBlock(32, 16);

				//execute cuda kernel
				convertToBlur<<<dimGrid, dimBlock>>>(d_inimg, d_outimg, bmpInfoHeader.biHeight, bmpInfoHeader.biWidth, elemSize);
				//copy computed gray data array from device to host
				hipMemcpy(outimg, d_outimg, sizeof(ubyte) * imageSize, hipMemcpyDeviceToHost);

				hipFree(d_outimg);
				hipFree(d_inimg);

				/***** write bmp *****/
				if((fp=fopen(argv[2], "wb"))==NULL) {
						fprintf(stderr, "Error : Failed to open file...₩n");
						return -1;
				}
				/*
				   palrgb = (RGBQUAD*)malloc(sizeof(RGBQUAD)*256);
				   for(int x = 0; x < 256; x++) {
				   palrgb[x].rgbBlue = palrgb[x].rgbGreen = palrgb[x].rgbRed = x;
				   palrgb[x].rgbReserved = 0;
				   }
				 */
				bmpInfoHeader.biBitCount = 24;
				bmpInfoHeader.SizeImage = imageSize;
				//bmpInfoHeader.biCompression = 0;
				//bmpInfoHeader.biClrUsed = 0;
				//bmpInfoHeader.biClrImportant = 0;
				//bmpHeader.bfOffBits = sizeof(BITMAPFILEHEADER) + sizeof(BITMAPINFOHEADER) + sizeof(RGBQUAD)*256;
				bmpHeader.bfSize = bmpInfoHeader.SizeImage;
				/* BITMAPFILEHEADER 구조체의 데이터 */
				fwrite(&bmpHeader, sizeof(BITMAPFILEHEADER), 1, fp);
				/* BITMAPINFOHEADER 구조체의 데이터 */
				fwrite(&bmpInfoHeader, sizeof(BITMAPINFOHEADER), 1, fp);
				//fwrite(palrgb, sizeof(RGBQUAD), 256, fp);
				//fwrite(inimg, sizeof(ubyte), imageSize, fp);
				fwrite(outimg, sizeof(ubyte), imageSize, fp);
				fclose(fp);
				free(inimg);
				free(outimg);

				printf("Success blur\n");
				return 0;
		}
