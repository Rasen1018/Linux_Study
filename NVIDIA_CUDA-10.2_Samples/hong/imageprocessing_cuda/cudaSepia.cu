#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h> /* USHRT_MAX 상수를 위해서 사용한다. */
#include <math.h>
#include <iostream>
#include "bmpHeader.h"
/* 이미지 데이터의 경계 검사를 위한 매크로 */
#define LIMIT_UBYTE(n) ((n)>UCHAR_MAX)?UCHAR_MAX:((n)<0)?0:(n)
#define widthbytes(bits) (((bits)+31)/32*4)
typedef unsigned char ubyte;
//Cuda kernel for converting RGB image into a GreyScale image

__global__ void convertToSepia(ubyte *rgb, ubyte *out, int rows, int cols, int elemSize) {
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	
	// Compute for only those threads which map directly to image grid
	if (col < cols && row < rows) {
		int rgb_offset = (row * cols + col) * elemSize;
		ubyte r = rgb[rgb_offset + 2];
		ubyte g = rgb[rgb_offset + 1];
		ubyte b = rgb[rgb_offset + 0];
		
		out[rgb_offset + 2] = LIMIT_UBYTE(r * 0.393f + g * 0.769f + b * 0.189f);
		out[rgb_offset + 1] = LIMIT_UBYTE(r * 0.349f + g * 0.686f + b * 0.168f);
		out[rgb_offset + 0] = LIMIT_UBYTE(r * 0.272f + g * 0.534f + b * 0.131f);
	}
}

int main(int argc, char** argv)
{
	FILE* fp;
	BITMAPFILEHEADER bmpHeader; /* BMP FILE INFO */
	BITMAPINFOHEADER bmpInfoHeader; /* BMP IMAGE INFO */
	//RGBQUAD *palrgb;
	ubyte *inimg, *outimg;
	if(argc != 3) {
		fprintf(stderr, "usage : %s input.bmp output.bmp\n", argv[0]);
		return -1;
	}
	/***** read bmp *****/
	if((fp=fopen(argv[1], "rb")) == NULL) {
		fprintf(stderr, "Error : Failed to open file...₩n");
		return -1;
	}
	/* BITMAPFILEHEADER 구조체의 데이터 */
	fread(&bmpHeader, sizeof(BITMAPFILEHEADER), 1, fp);
	/* BITMAPINFOHEADER 구조체의 데이터 */
	fread(&bmpInfoHeader, sizeof(BITMAPINFOHEADER), 1, fp);
	/* 트루 컬러를 지원하면 변환할 수 없다. */
	if(bmpInfoHeader.biBitCount != 24) {
		perror("This image file doesn't supports 24bit color\n");
		fclose(fp);
		return -1;
	}
	int elemSize = bmpInfoHeader.biBitCount/8.;
	int stride = bmpInfoHeader.biWidth * elemSize;
	//widthbytes(bits) (((bits)+31)/32*4)
	int imageSize = stride * bmpInfoHeader.biHeight;
	
	/* 이미지의 해상도(넓이 × 깊이) */
	printf("Resolution : %d x %d\n", bmpInfoHeader.biWidth, bmpInfoHeader.biHeight);
	printf("Bit Count : %d(%d:%d)\n", bmpInfoHeader.biBitCount, elemSize, stride);

	printf("Image Size : %d\n", imageSize);
	inimg = (ubyte*)malloc(sizeof(ubyte)*imageSize);
	outimg = (ubyte*)malloc(sizeof(ubyte)*imageSize);
	fread(inimg, sizeof(ubyte), imageSize, fp);
	fclose(fp);
			
	ubyte *d_inimg = NULL, *d_outimg = NULL;
	//allocate and initialize memory on device
	hipMalloc(&d_inimg, sizeof(ubyte) * imageSize);
	hipMalloc(&d_outimg, sizeof(ubyte) * imageSize);
	hipMemset(d_outimg, 0, sizeof(ubyte) * imageSize);
	//copy host rgb data array to device rgb data array
	hipMemcpy(d_inimg, inimg, sizeof(ubyte) * imageSize, hipMemcpyHostToDevice);
	
	//define block and grid dimensions
	const dim3 dimGrid((int)ceil((bmpInfoHeader.biWidth/32)), (int)ceil((bmpInfoHeader.biHeight)/16));
	const dim3 dimBlock(32, 16);

	//execute cuda kernel
	convertToSepia<<<dimGrid, dimBlock>>>(d_inimg, d_outimg, bmpInfoHeader.biHeight, bmpInfoHeader.biWidth, elemSize);
	
	//copy computed gray data array from device to host
	hipMemcpy(outimg, d_outimg, sizeof(ubyte) * imageSize, hipMemcpyDeviceToHost);

	hipFree(d_outimg);
	hipFree(d_inimg);
	
	/***** write bmp *****/
	if((fp=fopen(argv[2], "wb"))==NULL) {
		fprintf(stderr, "Error : Failed to open file...₩n");
		return -1;
	}
	/*
	palrgb = (RGBQUAD*)malloc(sizeof(RGBQUAD)*256);
	for(int x = 0; x < 256; x++) {
		palrgb[x].rgbBlue = palrgb[x].rgbGreen = palrgb[x].rgbRed = x;
		palrgb[x].rgbReserved = 0;
	}
	*/
	bmpInfoHeader.biBitCount = 24;
	bmpInfoHeader.SizeImage = imageSize;
	//bmpInfoHeader.biCompression = 0;
	//bmpInfoHeader.biClrUsed = 0;
	//bmpInfoHeader.biClrImportant = 0;
	//bmpHeader.bfOffBits = sizeof(BITMAPFILEHEADER) + sizeof(BITMAPINFOHEADER) + sizeof(RGBQUAD)*256;
	bmpHeader.bfSize = bmpInfoHeader.SizeImage;
	/* BITMAPFILEHEADER 구조체의 데이터 */
	fwrite(&bmpHeader, sizeof(BITMAPFILEHEADER), 1, fp);
	/* BITMAPINFOHEADER 구조체의 데이터 */
	fwrite(&bmpInfoHeader, sizeof(BITMAPINFOHEADER), 1, fp);
	//fwrite(palrgb, sizeof(RGBQUAD), 256, fp);
	//fwrite(inimg, sizeof(ubyte), imageSize, fp);
	fwrite(outimg, sizeof(ubyte), imageSize, fp);
	fclose(fp);
	free(inimg);
	free(outimg);
	return 0;
}