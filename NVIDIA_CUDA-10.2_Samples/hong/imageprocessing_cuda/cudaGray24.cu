#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h> /* USHRT_MAX 상수를 위해서 사용한다. */
#include <math.h>
#include <iostream>
#include "bmpHeader.h"
/* 이미지 데이터의 경계 검사를 위한 매크로 */
#define LIMIT_UBYTE(n) ((n)>UCHAR_MAX)?UCHAR_MAX:((n)<0)?0:(n)
#define widthbytes(bits) (((bits)+31)/32*4)
typedef unsigned char ubyte;

//Cuda kernel for converting RGB image into a GreyScale image

__global__ void convertToGray(ubyte *rgb, ubyte *gray, int height, int width, int elemSize) {
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;
		int size = width*elemSize;
		int offset = x*elemSize+(y*size);

		if( x < width && y < height){
				int r = rgb[offset + 2];
				int g = rgb[offset + 1];
				int b = rgb[offset + 0];
				gray[offset] = gray[offset+1] = gray[offset+2] = r * 0.299f + g * 0.587f + b * 0.114f;
		}

}
int main(int argc, char** argv)
{
		FILE* fp;
		BITMAPFILEHEADER bmpHeader; /* BMP FILE INFO */
		BITMAPINFOHEADER bmpInfoHeader; /* BMP IMAGE INFO */
		ubyte *inimg, *outimg;
		if(argc != 3) {
				fprintf(stderr, "usage : %s input.bmp output.bmp\n", argv[0]);
				return -1;
		}
		/***** read bmp *****/
		if((fp=fopen(argv[1], "rb")) == NULL) {
				fprintf(stderr, "Error : Failed to open file...₩n");
				return -1;
		}
		/* BITMAPFILEHEADER 구조체의 데이터 */
		fread(&bmpHeader, sizeof(BITMAPFILEHEADER), 1, fp);
		/* BITMAPINFOHEADER 구조체의 데이터 */
		fread(&bmpInfoHeader, sizeof(BITMAPINFOHEADER), 1, fp);
		/* 트루 컬러를 지원하면 변환할 수 없다. */
		if(bmpInfoHeader.biBitCount != 24) {
				perror("This image file doesn't supports 24bit color\n");
				fclose(fp);
				return -1;
		}
		int elemSize = bmpInfoHeader.biBitCount/8.;
		int stride = bmpInfoHeader.biWidth * elemSize;
		int imageSize = stride * bmpInfoHeader.biHeight;
		/* 이미지의 해상도(넓이 × 깊이) */

		printf("Resolution : %d x %d\n", bmpInfoHeader.biWidth, bmpInfoHeader.biHeight);
		printf("Bit Count : %d(%d:%d)\n", bmpInfoHeader.biBitCount, elemSize, stride);
		printf("Image Size : %d\n", imageSize);
		inimg = (ubyte*)malloc(sizeof(ubyte)*imageSize);
		outimg = (ubyte*)malloc(sizeof(ubyte)*imageSize);
		fread(inimg, sizeof(ubyte), imageSize, fp);
		fclose(fp);

		ubyte *d_inimg = NULL, *d_outimg = NULL;

		hipMalloc(&d_inimg, sizeof(ubyte) * imageSize);
		hipMalloc(&d_outimg, sizeof(ubyte) * imageSize);

		hipMemset(d_outimg, 0, sizeof(ubyte) * imageSize);
		hipMemcpy(d_inimg, inimg, sizeof(ubyte) * imageSize, hipMemcpyHostToDevice);

		const dim3 dimGrid((int)ceil((bmpInfoHeader.biWidth/32)), (int)ceil((bmpInfoHeader.biHeight)/16));
		const dim3 dimBlock(32, 16);

		convertToGray<<<dimGrid, dimBlock>>>(d_inimg, d_outimg, bmpInfoHeader.biHeight, bmpInfoHeader.biWidth, elemSize);
		hipMemcpy(outimg, d_outimg, sizeof(ubyte) * imageSize, hipMemcpyDeviceToHost);

		hipFree(d_outimg);
		hipFree(d_inimg);

		/***** write bmp *****/
		if((fp=fopen(argv[2], "wb"))==NULL) {
				fprintf(stderr, "Error : Failed to open file...₩n");
				return -1;
		}

		bmpInfoHeader.biBitCount = 24;
		/* BITMAPFILEHEADER 구조체의 데이터 */
		fwrite(&bmpHeader, sizeof(BITMAPFILEHEADER), 1, fp);
		/* BITMAPINFOHEADER 구조체의 데이터 */
		fwrite(&bmpInfoHeader, sizeof(BITMAPINFOHEADER), 1, fp);
		fwrite(outimg, sizeof(ubyte), imageSize, fp);

		fclose(fp);

		free(inimg);
		free(outimg);
		return 0;
}
